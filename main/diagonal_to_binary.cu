#include <CLI11/CLI11.hpp>
#include <SNIG/utility/reader.hpp>


int main(int argc, char* argv[]) {

  // automaticlly generate digonal test data to binary file.

  // usage: ./diagonal_to_binary
  //          --neurons(-n) 1024, 4096, or 16384
  //          --layers(-l)  120, 480, or 1920
  //          --weight_path(-w) output path of weight
  //          --input_path(-i)  output path of input
  //          --golden_path(-g) output path of golden
  //          --golden_all  Convert all golden files less or equal to  --layers

  // example1:
  //        ./diagonal_to_binary 
  // example2:
  //        ./diagonal_to_binary -n 1024 -l 1920 -w ../sample_data/test/weight/neuron1024/ -i ../sample_data/test/MNIST/ -g ../sample_data/test/MNIST/ --golden_all true

  // COL_BLK, N_SLAB would be caculated automatically, based on GPU architecture.

  CLI::App app{"Digonal_test_data_Generator"};

  size_t num_neurons_per_layer = 1024;
  app.add_option("-n, --neurons", 
    num_neurons_per_layer, 
    "select number of neurons, default is 1024");

  size_t num_layers = 120;
  app.add_option("-l, --layers", 
    num_layers, 
    "select number of layers, default is 120");

  bool golden_all = true;
  app.add_option("--golden_all", 
    golden_all, 
    "this would convert all golden files with the same neurons. Otherwise only specific num_layers and num_neurons would be converted. Default is true");

  std::fs::path weight_path("../sample_data/test/weight/neuron1024/");
  app.add_option("-w, --weight_path", 
    weight_path, 
    "select directory of weights. Output binary files would also be generated here. Default is ../sample_data/test/weight/neuron1024/");

  std::fs::path input_path("../sample_data/test/MNIST/");
  app.add_option("-i, --input_path", 
    input_path, 
    "select input path. Output binary files would also be generated here. Default is ../sample_data/test/MNIST/");

  std::fs::path golden_path("../sample_data/test/MNIST/");
  app.add_option("-g, --golden_path", 
    golden_path, 
    "select golden path. Output binary files would also be generated here. Default is ../sample_data/test/MNIST/");

  CLI11_PARSE(app, argc, argv);

  size_t COL_BLK;
  size_t N_SLAB;

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  size_t max_num_per_block = props.sharedMemPerBlock / sizeof(float);

  if(num_neurons_per_layer <= max_num_per_block) {
    COL_BLK = num_neurons_per_layer;
  }
  else{
    int max_divisor = 2;
    while((num_neurons_per_layer % max_divisor != 0) || (max_num_per_block < (num_neurons_per_layer / max_divisor))) {
      ++max_divisor;
    }
    COL_BLK = num_neurons_per_layer / max_divisor;
  }

  N_SLAB = num_neurons_per_layer / COL_BLK; 

  std::cout << "Transforming weight files...\n";

  snig::diagonal_to_binary_file<float>(
    weight_path,
    num_layers,
    num_neurons_per_layer,
    num_neurons_per_layer,
    COL_BLK,
    N_SLAB
  ); 

  std::cout << "Transforming input files...\n";

  snig::diagonal_to_binary_file<float>(
    input_path,
    60000,
    num_neurons_per_layer
  );

  std::cout << "Transforming golden files...\n";

  if(!golden_all){
    snig::diagonal_to_binary_file(
      golden_path,
      num_neurons_per_layer,
      num_layers,
      60000
    );
  }
  else{
    for(int i = 120; i <= num_layers; i *= 4){
      snig::diagonal_to_binary_file(
        golden_path,
        num_neurons_per_layer,
        i,
        60000
      );
    }
  }
}
